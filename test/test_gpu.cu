
#include <hip/hip_runtime.h>

void __global__ kernel_test(float *data, int N){
  int tx = threadIdx.x + blockIdx.x*blockDim.x;
  if (tx < N)
    data[tx] = tx;  
}

void gpu_test(float *data, int N){
  float *d_data = NULL;	
	hipMalloc(&d_data, N*sizeof(float));	
	kernel_test<<<1,N>>>(d_data, N);
	hipMemcpy(data, d_data, N*sizeof(float), hipMemcpyDeviceToHost);
  if (d_data)
    hipFree(d_data);
}